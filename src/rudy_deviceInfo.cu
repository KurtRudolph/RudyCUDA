/**
  * @file rudy_deviceInfo.cu
	* @author Kurt Robert Rudolph
	* @description This file defines functions which 
	* gather info about a device
	*/
#include "rudy_deviceInfo.h"

hipDeviceProp_t ** rudy_deviceInfo_gather(void){
	int i;
	int deviceCount;
	HANDLE_ERROR( hipGetDeviceCount( &deviceCount));
	hipDeviceProp_t ** devicePropertiesArray = (hipDeviceProp_t**) malloc (sizeof (hipDeviceProp_t*) * deviceCount);
	for (i = 0; i< deviceCount; i++) {
		devicePropertiesArray[i] = (hipDeviceProp_t*) malloc (sizeof (hipDeviceProp_t));	
		HANDLE_ERROR( hipGetDeviceProperties (devicePropertiesArray[i], i));
	}
	printf("\n rud_deviceInfo_gather()\n");

	return devicePropertiesArray;
}
